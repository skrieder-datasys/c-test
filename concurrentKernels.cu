#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <stdio.h>
#include <cutil_inline.h>
#include <shrUtils.h>
#include <shrQATest.h>

const char *sSDKsample = "concurrentKernels";

// This is a kernel that does no real work but runs at least for a specified number of clocks
__global__ void clock_block(clock_t* d_o, clock_t clock_count)
{ 
	clock_t start_clock = clock();
	
	clock_t clock_offset = 0;

	while( clock_offset < clock_count ) {
		clock_offset = clock() - start_clock;
	}

	d_o[0] = clock_offset;
}

// Single warp reduction kernel
__global__ void sum(clock_t* d_clocks, int N)
{
	__shared__ clock_t s_clocks[32];

	clock_t my_sum = 0;

	for( int i = threadIdx.x; i < N; i+= blockDim.x ) {
		my_sum += d_clocks[i];
	}

	s_clocks[threadIdx.x] = my_sum;
	syncthreads();	

	for( int i=16; i>0; i/=2) {
		if( threadIdx.x < i ) {
			s_clocks[threadIdx.x] += s_clocks[threadIdx.x + i];
		}
		syncthreads();	
	}	

	d_clocks[0] = s_clocks[0];
}

int main(int argc, char **argv)
{
    int nkernels = 16;               // number of concurrent kernels
    int nstreams = nkernels + 1;    // use one more stream than concurrent kernel
    int nbytes = nkernels * sizeof(clock_t);   // number of data bytes
    float kernel_time = 10; // time the kernel should run in ms
//    float elapsed_time;   // timing variables
    int cuda_device = 0;

    shrQAStart(argc, argv); 

    // get number of kernels if overridden on the command line
    if (cutCheckCmdLineFlag(argc, (const char **)argv, "nkernels")) {
        cutGetCmdLineArgumenti(argc, (const char **)argv, "nkernels", &nkernels);
        nstreams = nkernels + 1;
    }

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    cuda_device = cutilChooseCudaDevice(argc, argv);

    hipDeviceProp_t deviceProp;
    cutilSafeCall( hipGetDevice(&cuda_device));	

    cutilSafeCall( hipGetDeviceProperties(&deviceProp, cuda_device) );

    // allocate host memory
    clock_t *a = 0;                     // pointer to the array data in host memory
    cutilSafeCall( hipHostMalloc((void**)&a, nbytes) ); 

    // allocate device memory
    clock_t *d_a = 0;             // pointers to data and init value in the device memory
    cutilSafeCall( hipMalloc((void**)&d_a, nbytes) );

    // allocate and initialize an array of stream handles
    hipStream_t *streams = (hipStream_t*) malloc(nstreams * sizeof(hipStream_t));
    for(int i = 0; i < nstreams; i++)
        cutilSafeCall( hipStreamCreate(&(streams[i])) );

    hipEvent_t *kernelEvent;
    kernelEvent = (hipEvent_t*) malloc(nkernels * sizeof(hipEvent_t));
    for(int i = 0; i < nkernels; i++)
        cutilSafeCall( hipEventCreateWithFlags(&(kernelEvent[i]), hipEventDisableTiming) );

    //////////////////////////////////////////////////////////////////////
    // time execution with nkernels streams
    clock_t total_clocks = 0;
    clock_t time_clocks = kernel_time * deviceProp.clockRate;
    //shrLog("Time Clocks time is: %d", time_clocks);	
    // hipEventRecord(start_event, 0);
    // queue nkernels in separate streams and record when they are done
    for( int i=0; i<nkernels; ++i)
    {
        clock_block<<<1,1,0,streams[i]>>>(&d_a[i], time_clocks );
        total_clocks += time_clocks;
        cutilSafeCall( hipEventRecord(kernelEvent[i], streams[i]) );
	
        // make the last stream wait for the kernel event to be recorded
        cutilSafeCall( hipStreamWaitEvent(streams[nstreams-1], kernelEvent[i],0) );
    }

    // queue a sum kernel and a copy back to host in the last stream. 
    // the commands in this stream get dispatched as soon as all the kernel events have been recorded
    sum<<<1,32,0,streams[nstreams-1]>>>(d_a, nkernels);
    cutilSafeCall( hipMemcpyAsync(a, d_a, sizeof(clock_t), hipMemcpyDeviceToHost, streams[nstreams-1]) );
 
    // at this point the CPU has dispatched all work for the GPU and can continue processing other tasks in parallel

    // in this sample we just wait until the GPU is done

    // release resources
    for(int i = 0; i < nkernels; i++) {
        hipStreamDestroy(streams[i]); 
        hipEventDestroy(kernelEvent[i]);
    }
    free(streams);
    free(kernelEvent);

    hipHostFree(a);
    hipFree(d_a);

    cutilDeviceReset();
  return 0;    
}